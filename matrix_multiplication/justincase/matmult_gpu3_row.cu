extern "C"
{


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 16

__global__ 
void
matmultgpu3_colwise(int m, int n, int k, double *A, double *B, double *C) {
    // Bad one
 double Cvalue1 = 0.0, 
        Cvalue2 = 0.0;

  int col = 2*(blockIdx.x*blockDim.x+threadIdx.x);
  int row=blockIdx.y*blockDim.y+threadIdx.y;
  
  int e;

  if ((row < m) && (col < (n - 1))) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[row*k+e]*B[e*n+col+1];
    }
        
    C[row*n+col]=Cvalue1;
    C[row*n+col+1]=Cvalue2;
  }

  else if ((row < m) && (col == (n - 1))) {
    for(e=0;e<k;++e)
        Cvalue1+=A[row*k+e]*B[e*n+col];
        
        C[row*n+col]=Cvalue1;
  }
	
}

__global__ 
void
matmultgpu3_rowwise(int m, int n, int k, double *A, double *B, double *C) {
    // This is the good one!!!
    	
 double Cvalue1 = 0.0, 
        Cvalue2 = 0.0;

  int col = blockIdx.x*blockDim.x+threadIdx.x;
  int row = 2*(blockIdx.y*blockDim.y+threadIdx.y);
  
  int e;

  if ((row < m-1) && (col < n)) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[(row+1)*k+e]*B[e*n+col];
    }
        
    C[row*n+col]=Cvalue1;
    C[(row+1)*n+col]=Cvalue2;
  }

  else if ((row == m -1) && (col < n)) {
    for(e=0;e<k;++e)
        Cvalue1+=A[row*k+e]*B[e*n+col];
        
        C[row*n+col]=Cvalue1;
  }
	
}



void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C){

  double *d_A, *d_B, *d_C;

  int blocky;
  int sizeA = m * k *sizeof(double);
  int sizeB = k * n *sizeof(double);
  int sizeC = m * n *sizeof(double);

  // Allocate memory on the device
  hipMalloc((void**)&d_A, sizeA);
  hipMalloc((void**)&d_B, sizeB);
  hipMalloc((void**)&d_C, sizeC);

  // Copy the values over
  hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

  // Declare the number of threads
  dim3 numOfThreadsPerBlock;
  numOfThreadsPerBlock.x = BLOCK_SIZE;
  numOfThreadsPerBlock.y = BLOCK_SIZE;


  // Initializing for colwise
//   blocky = (n+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.x);
//   dim3 numOfBlocks;
//     numOfBlocks.x = (blocky+1)/2;
//   numOfBlocks.y = (m+numOfThreadsPerBlock.y-1)/(numOfThreadsPerBlock.y);

  // Initializing for rowwise
  blocky = (m+numOfThreadsPerBlock.y-1)/(numOfThreadsPerBlock.y);
  dim3 numOfBlocks;
    numOfBlocks.x = (n+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.x);
  numOfBlocks.y = (blocky+1)/2;

  matmultgpu3_rowwise<<<numOfBlocks, numOfThreadsPerBlock>>>(m, n, k, d_A, d_B, d_C);
  hipDeviceSynchronize();

  hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
}