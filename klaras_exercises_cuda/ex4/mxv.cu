
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mxv(int m, int n, double *a, double *b, double *c){
	printf("Have we at least entered the function?\n");
	int index,j;
	index = threadIdx.x + blockIdx.x*blockDim.x;
	double sum;
	sum = 0.0;
	if(index<m){
		for (j=0; j<n; j++){
		    sum += a[m*j + index]*b[j];
		}
	c[index] = sum;
	}
} 